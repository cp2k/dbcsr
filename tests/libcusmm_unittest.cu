#include "hip/hip_runtime.h"
/*****************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations        *
 *  Copyright (C) 2000 - 2018  CP2K developers group                         *
 *****************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "acc/libsmm_acc/libcusmm/libcusmm_benchmark.h"
#include "acc/libsmm_acc/libcusmm/libcusmm.h"

/****************************************************************************\
 \brief Checks correctness of every libcusmm kernel and measures its performance.
\****************************************************************************/

int main(int argc, char** argv){

    printf("In libcusmm unit test (toy)\n"); 
    KernelLauncher launcher = libcusmm_process_d;

    char buffer[1000];
    char * kernel_descr[1] = {buffer};

    std::vector<int> v;

    /*int const p_min = 4;
    int const p_max = 32;
    for(size_t m=p_min; m<=p_max; m++){
        for(size_t n=p_min; n<=p_max; n++){
            for(size_t k=p_min; k<=p_max; k++){
                v.push_back(m); v.push_back(n); v.push_back(k);
            }
        }
     }*/

    v.push_back(6); v.push_back(6); v.push_back(6);
    v.push_back(6); v.push_back(6); v.push_back(64);
    v.push_back(6); v.push_back(64); v.push_back(6);
    v.push_back(64); v.push_back(6); v.push_back(6);
    v.push_back(6); v.push_back(64); v.push_back(64);
    v.push_back(64); v.push_back(6); v.push_back(64);
    v.push_back(64); v.push_back(64); v.push_back(6);
    v.push_back(64); v.push_back(64); v.push_back(64);
    
    int n_blocksizes = v.size()/3;
    const int *blocksizes = &v[0];
    printf("# Libcusmm has %d blocksizes compiled in...\n", n_blocksizes);

    int max_m=0, max_n=0, max_k=0;
    for(int i=0; i<n_blocksizes; i++){
        max_m = std::max(max_m, blocksizes[3*i + 0]);
        max_n = std::max(max_n, blocksizes[3*i + 1]);
        max_k = std::max(max_k, blocksizes[3*i + 2]);
    }

    libcusmm_benchmark_t* handle;
    libcusmm_benchmark_init(&handle, false, max_m, max_n, max_k);
    printf("Initialized benchmarking\n");

    int errors = 0;
    for(int i=0; i<n_blocksizes; i++){
        int m = blocksizes[3*i + 0];
        int n = blocksizes[3*i + 1];
        int k = blocksizes[3*i + 2];
        sprintf(buffer, "%d x %d x %d", m, n, k);
        errors += libcusmm_benchmark(handle, m, n, k, 1, &launcher, kernel_descr);
    }
    libcusmm_benchmark_finalize(handle);

    printf("# Done, found %d errors.\n", errors);
    return(errors);
}

//EOF
