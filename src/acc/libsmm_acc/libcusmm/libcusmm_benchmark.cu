#include "hip/hip_runtime.h"
/*****************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations        *
 *  Copyright (C) 2000 - 2018  CP2K developers group                         *
 *****************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include "libcusmm_benchmark.h"
#include "parameters.h"
#include "parameters_utils.h"

//===========================================================================
// Allocate memory and cuda events
void libcusmm_benchmark_init(libcusmm_benchmark_t** handle, bool tune_mode,
                             int max_m, int max_n, int max_k){

    libcusmm_benchmark_t* h = (libcusmm_benchmark_t*) malloc(sizeof(libcusmm_benchmark_t));
    *handle = h;

    h-> tune_mode = tune_mode;

    if(h->tune_mode){
       h->n_a = 10000;
       h->n_b = 10000;
       h->n_c = 1000;
       h->n_stack = 16005;
//       h->n_a = 100;
//       h->n_b = 100;
//       h->n_c = 10;
//       h->n_stack = 18;
    }else{
       h->n_a = 100;
       h->n_b = 100;
       h->n_c = 10;
       h->n_stack = 100;
    }

    h->max_m = max_m;
    h->max_n = max_n;
    h->max_k = max_k;

    h->mat_a = (double*) malloc(h->n_a * max_m * max_k * sizeof(double));
    h->mat_b = (double*) malloc(h->n_b * max_k * max_n * sizeof(double));
    h->mat_c = (double*) malloc(h->n_c * max_m * max_n * sizeof(double));
    h->stack = (int*) malloc(h->n_stack * 3 * sizeof(int));

    hipMalloc(&h->d_mat_a, h->n_a * max_m * max_k * sizeof(double));
    hipMalloc(&h->d_mat_b, h->n_b * max_k * max_n * sizeof(double));
    hipMalloc(&h->d_mat_c, h->n_c * max_m * max_n * sizeof(double));
    hipMalloc(&h->d_stack, h->n_stack * 3 * sizeof(int));

    hipEventCreateWithFlags(&h->t_start, hipEventDefault);
    hipEventCreateWithFlags(&h->t_stop, hipEventDefault);

    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess){
      printf("libcusmm_benchmark_init: %s\n", hipGetErrorString(hipError_t));
      exit(1);
    }
}


//===========================================================================
// Free memory and cuda events
void libcusmm_benchmark_finalize(libcusmm_benchmark_t* handle){
    hipEventDestroy(handle->t_stop);
    hipEventDestroy(handle->t_start);
    hipFree(handle->d_stack);
    hipFree(handle->d_mat_c);
    hipFree(handle->d_mat_b);
    hipFree(handle->d_mat_a);
    free(handle->stack);
    free(handle->mat_c);
    free(handle->mat_b);
    free(handle->mat_a);
    free(handle);
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess){
      printf("libcusmm_benchmark_finalize: %s\n", hipGetErrorString(hipError_t));
      exit(1);
    }
}


//===========================================================================
// initialize matrix
void matInit(double* mat, int mat_n, int x, int y, int seed){

 double *m = mat;

 for(int n=0; n<mat_n; n++){
   for(int j=0; j<y; j++) {
     for(int i=0; i<x; i++, m++) {
     *m = (double) j*x + i + n + seed;
     //printf("matrix [%d, %d]=%g\n", i, j, *m);
     }
   }
 }

}


//===========================================================================
// initialize the task list ("stack" in CP2K lingo)
// for each of the result matrices we have a random number
void stackInit(int *stack, int n_stack, int n_c, double* mat_c,
               int n_a, double * mat_a, int n_b, double* mat_b,
               int mat_m, int mat_n, int mat_k){

  if(n_stack < n_c){
    printf("Error: n_stack < n_c\n");
    exit(1);
  }

  // on average, we have n_avg matrix products contributing to a result mat_c
  int n_avg = n_stack / n_c;

  int n_imbalance = std::max(1, n_avg-4);

  int c = 0;
  int n_top = 0;
  int p = 0;

 int* s = stack;
  while( p < n_stack ){
    if(c >= n_c) c = n_c-1;

    n_top += n_avg + (rand() % (2*n_imbalance) - n_imbalance);
    if(n_top > n_stack) n_top = n_stack;

    for(;p < n_top; p++){
     int a = rand() % n_a;
     int b = rand() % n_b;

     *s++ =  a * mat_m * mat_k + 1;        // A_src
     *s++ =  b * mat_k * mat_n + 1;        // B_src
     *s++ =  c * mat_m * mat_n + 1;        // C_dst
    }
    c++;
 }
}


//===========================================================================
void stackCalc(int* stack, int n_stack, double* mat_c, double *mat_a, double* mat_b,
               int mat_m, int mat_n, int mat_k){

  for(int s=0; s<n_stack; s++){
     int a_base = stack[3 * s    ] - 1;
     int b_base = stack[3 * s + 1] - 1;
     int c_base = stack[3 * s + 2] - 1;

     for(int n=0; n<mat_n; n++){
       for(int m=0; m<mat_m; m++){
         double res = 0.;
         for(int k=0; k<mat_k; k++){
          int a_ind = k * mat_m + m;


//         // initialize with non-transpose matrix
//         int b_ind = n * mat_k + k;
//         res += mat_a[a_base + a_ind] * mat_b[b_base + b_ind];

          // initialize with transpose matrix
          int b_ind = k * mat_n + n;
          res += mat_a[a_base + a_ind] * mat_b[b_base + b_ind];
         }
         int c_ind = n * mat_m +  m;
         mat_c[c_base + c_ind] += res;
       }
     }
  }

}


//===========================================================================
double checkSum(double* mat_c, int n_c, int mat_m, int mat_n){
   double res = 0;
   for(int i=0; i<n_c * mat_m * mat_n; i++){
     res += mat_c[i];
   }
   return res;
}


//===========================================================================
//Removes special symbols so that the output is usefull for awk and gnuplot.
static void clean_string(char* str_in, char* str_out){
    for(int i=0; i<1000 ; i++){
        if(str_in[i] == '=' || str_in[i] == ',' || str_in[i] == '(' || str_in[i] == ')'){
            str_out[i] = ' ';
         }else{
             str_out[i] = str_in[i];
         }
         if(str_in[i] == 0)
             break;
    }
}


//===========================================================================
int libcusmm_benchmark(libcusmm_benchmark_t* h,
                       int mat_m, int mat_n, int mat_k,
                       int nkernels, KernelLauncher* launchers, char ** kernel_descr){

 if(mat_m > h->max_m || mat_n > h->max_n || mat_k > h->max_k){
     printf("libcusmm_benchmark: got handle with too few resources\n");
     exit(1);
 }
 int h_mnk = hash(mat_m, mat_n, mat_k); 
 std::vector<int> blocksizes; 
 get_libcusmm_triplets(blocksizes, ht); 
 auto it = std::find(std::begin(blocksizes), std::end(blocksizes), h_mnk); 
 if(it == std::end(blocksizes) && !h->tune_mode){
     printf("Triplet %i x %i x %i is not defined in libcusmm\n", mat_m, mat_n, mat_k);
     exit(1);
 }

 int n_iter = 1;
 int n_warm = 1;
 if(h->tune_mode){ // for larger matrices few iteration give enough statistics
//     n_iter = max(3, 1250/(mat_m * mat_n * mat_k));
     n_iter = max(3, 12500/(mat_m * mat_n * mat_k));
     n_warm = min(3, n_iter);
 }

 hipStream_t stream; 
 hipStreamCreateWithFlags(&stream, hipStreamDefault);

 int error_counter = 0;
 int best_kernel = -1;
 double best_gflops = 0.0;
 double sumCPU, sumGPU;
 float t_duration;
 char descr[1000], msg_prefix[100]="";
 hipError_t hipError_t;

 memset(h->mat_c, 0, h->n_c * mat_m * mat_n * sizeof(double));
 matInit(h->mat_a, h->n_a, mat_m, mat_k, 42);
 matInit(h->mat_b, h->n_b, mat_k, mat_n, 24);

 if(h->tune_mode)
     printf("Initializing ...\n");
 stackInit(h->stack, h->n_stack, h->n_c, h->mat_c, h->n_a, h->mat_a, h->n_b, h->mat_b, mat_m, mat_n, mat_k);

 // Actually, we would have to calculate the stack n_iter times.
 // We cheat by simply scaling the results of a single stack calulcation.
 stackCalc(h->stack, h->n_stack, h->mat_c, h->mat_a, h->mat_b, mat_m, mat_n, mat_k);
 for(int i=0 ; i < h->n_c*mat_m*mat_n ; i++)
     h->mat_c[i] *= n_iter;

 sumCPU =  checkSum(h->mat_c, h->n_c, mat_m, mat_n);

 hipMemcpy(h->d_mat_a, h->mat_a, h->n_a * mat_m * mat_k * sizeof(double), hipMemcpyHostToDevice);
 hipMemcpy(h->d_mat_b, h->mat_b, h->n_b * mat_k * mat_n * sizeof(double), hipMemcpyHostToDevice);
 hipMemcpy(h->d_stack, h->stack, h->n_stack * 3 * sizeof(int), hipMemcpyHostToDevice);
 //d_mat_c get's zeroed after warmup run

 for(int ikern=0; ikern < nkernels; ikern++){
    //warmup run (more often if n_iter is small)
    for(int i=0; i<n_warm; i++)
        launchers[ikern](h->d_stack, h->n_stack, stream, mat_m, mat_n, mat_k, h->d_mat_a, h->d_mat_b, h->d_mat_c);
    hipMemset(h->d_mat_c, 0, h->n_c * mat_m * mat_n * sizeof(double));

    hipEventRecord(h->t_start, stream);

    for(int i=0; i<n_iter; i++)
        launchers[ikern](h->d_stack, h->n_stack, stream, mat_m, mat_n, mat_k, h->d_mat_a, h->d_mat_b, h->d_mat_c);

    hipEventRecord(h->t_stop, stream);
    hipEventSynchronize(h->t_stop);
    hipEventElapsedTime(&t_duration, h->t_start, h->t_stop);

    hipMemcpy(h->mat_c, h->d_mat_c, h->n_c * mat_m * mat_n * sizeof(double), hipMemcpyDeviceToHost);

    clean_string(kernel_descr[ikern], descr);

    if(h->tune_mode)
        sprintf(msg_prefix, "params %d / %d\n",ikern+1, nkernels);

    hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess){
      printf("%sERROR %s cuda_error: %s\n", msg_prefix, descr, hipGetErrorString(hipError_t));
      error_counter++;
      continue;
    }

    sumGPU =  checkSum(h->mat_c, h->n_c, mat_m, mat_n);
    if(sumGPU != sumCPU){
        printf("%sERROR %s checksum_diff: %g\n",msg_prefix, descr, sumGPU-sumCPU);
        error_counter++;
        continue;
    }

    if(h->tune_mode){
       double gflops = ((double) n_iter * h->n_stack * mat_m * mat_n * mat_k * 2 / (1e9))/(t_duration * 1e-3);
       printf("%sOK %s GFlop/s %g\n", msg_prefix, descr, gflops);
       if(best_gflops < gflops){
           best_gflops = gflops;
           best_kernel = ikern;
       }
    }else{
       printf("%sOK %s\n", msg_prefix, descr);
    }
 }

 if(h->tune_mode){
    printf("\n\n");
    if(best_kernel > -1){
        printf("WINNER: %d %s , # %g GFlop/s \n", best_kernel+1, kernel_descr[best_kernel], best_gflops);
    }else{
       printf("WINNER: None\n");
    }
    printf("Number of errors: %d\n", error_counter);
    hipDeviceReset();
 }

 // cleanup
 return(error_counter);
}

//EOF
