/*****************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations        *
 *  Copyright (C) 2000 - 2018  CP2K developers group                         *
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../include/acc.h"

#ifdef __CUDA_PROFILING
#include <nvToolsExtCudaRt.h>
#endif

#define CUDA_SAFE_CALL(name, x)                                   \
  do {                                                            \
    hipError_t result = x;                                          \
    if (result != hipSuccess) {                                 \
      const char *msg;                                            \
      hipDrvGetErrorName(result, &msg);                               \
      printf("\nerror: %s failed with error %s\n",                \
             name, msg);                                          \
      exit(1);                                                    \
    }                                                             \
  } while(0)


/****************************************************************************/
extern "C" int acc_init(){
  // Driver boilerplate
  CUDA_SAFE_CALL("hipInit", hipInit(0));
  hipDevice_t cuDevice; 
  CUDA_SAFE_CALL("hipDeviceGet", hipDeviceGet(&cuDevice, 0));
  hipCtx_t ctx;
  CUDA_SAFE_CALL("hipDevicePrimaryCtxRetain", hipDevicePrimaryCtxRetain(&ctx, cuDevice));
  CUDA_SAFE_CALL("hipCtxPushCurrent", hipCtxPushCurrent(ctx));
  return 0;
}

/****************************************************************************/
extern "C" int acc_finalize(){
  // Release driver resources
  hipCtx_t ctx;
  CUDA_SAFE_CALL("hipCtxGetCurrent", hipCtxGetCurrent(&ctx)); 
  CUDA_SAFE_CALL("hipCtxPopCurrent", hipCtxPopCurrent(&ctx)); 
  hipDevice_t cuDevice;
  CUDA_SAFE_CALL("hipDeviceGet", hipDeviceGet(&cuDevice, 0));
  CUDA_SAFE_CALL("hipDevicePrimaryCtxRelease", hipDevicePrimaryCtxRelease(cuDevice));
  return 0;
}

