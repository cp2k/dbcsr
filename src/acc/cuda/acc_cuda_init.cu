/*****************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations        *
 *  Copyright (C) 2000 - 2017  CP2K developers group                         *
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../include/acc.h"

#ifdef __CUDA_PROFILING
#include <nvToolsExtCudaRt.h>
#endif

#define CUDA_SAFE_CALL(name, x)                                   \
  do {                                                            \
    hipError_t result = x;                                          \
    if (result != hipSuccess) {                                 \
      const char *msg;                                            \
      hipDrvGetErrorName(result, &msg);                               \
      printf("\nerror: %s failed with error %s",                  \
             name, msg);                                          \
      exit(1);                                                    \
    }                                                             \
  } while(0)


/****************************************************************************/
extern "C" int acc_drv_init(){
  // Driver boilerplate
  CUDA_SAFE_CALL("hipInit", hipInit(0));
  CUDA_SAFE_CALL("hipDeviceGet", hipDeviceGet(&cuDevice, 0));
  CUDA_SAFE_CALL("hipCtxGetCurrent", hipCtxGetCurrent(&context));
  if(context == NULL){
    printf("hipCtxGetCurrent error: no context is bound to the calling CPU thread");
    return -1;
  }
  return 0;
}


